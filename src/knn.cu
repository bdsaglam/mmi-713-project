#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath> // For fabsf
#include <iostream>
#include <stdlib.h>
#include "common.h"
#include "sorting.h"

#define DEBUG 1

// CPU

void computeL1DistanceCPU(float *documents, float *queries, float *output, int D, int N, int Q) {
    for (int q = 0; q < Q; ++q) {
        for (int n = 0; n < N; ++n) {
            for (int d = 0; d < D; ++d) {
                int docIndex = n * D + d;
                int queryIndex = q * D + d;
                int outputIndex = (q * N + n) * D + d;
                output[outputIndex] = fabsf(queries[queryIndex] - documents[docIndex]);
            }
        }
    }
}

// Function to sum over the last dimension
void sumOverLastDim(float *h_distances, float *h_output, int D, int N, int Q) {
    for (int q = 0; q < Q; ++q) {
        for (int n = 0; n < N; ++n) {
            float sum = 0.0f;
            for (int d = 0; d < D; ++d) {
                int index = (q * N + n) * D + d;
                sum += h_distances[index];
            }
            int outputIndex = q * N + n;
            h_output[outputIndex] = sum;
        }
    }
}

// GPU

__global__ void computeL1Distance(float *documents, float *queries, float *output, int D, int N, int Q) {
    // Calculate the thread's unique ID
    int qIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int nIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int dIndex = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure the thread ID is within the bounds of the queries, documents, and dimensions
    if (qIndex < Q && nIndex < N && dIndex < D) {
        int docIndex = nIndex * D + dIndex;
        int queryIndex = qIndex * D + dIndex;
        int outputIndex = (qIndex * N + nIndex) * D + dIndex;
        output[outputIndex] = fabsf(queries[queryIndex] - documents[docIndex]);
    }
}

// GPU kernel for summing distance values over the last dimension of a 3D array (QxNxD) flattened in memory
__global__ void sumOverLastDimKernel(float *g_idata, float *g_odata, int D, int N, int Q) {
    extern __shared__ float sdata[];

    // Calculate the global and shared memory indices
    unsigned int tid = threadIdx.x;
    unsigned int q = blockIdx.z;
    unsigned int n = blockIdx.y;
    unsigned int i = q * (N * D) + n * D + tid;

    // Load data into shared memory
    if (tid < D) {
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        g_odata[q * N + n] = sdata[0];
    }
}


int main() {
    // Example dimensions
    int D = 512;   // Dimensionality
    int N = 100;  // Number of documents
    int Q = 4;   // Number of queries
    int K = 10; // Number of matches to return

    // Allocate host memory
    float *h_documents = (float *)malloc(N * D * sizeof(float));
    float *h_queries = (float *)malloc(Q * D * sizeof(float));
    float *h_results = (float *)malloc(Q * N * sizeof(float));

    // Initialize data with random values
    // srand(time(NULL));
    randomInit(h_documents, N, D);
    randomInit(h_queries, Q, D);

    // Allocate device memory
    float *d_documents, *d_queries, *d_distances, *d_results;
    hipMalloc(&d_documents, N * D * sizeof(float));
    hipMalloc(&d_queries, Q * D * sizeof(float));
    hipMalloc(&d_distances, Q * N * D * sizeof(float));
    hipMalloc(&d_results, Q * N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_documents, h_documents, N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_queries, h_queries, Q * D * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid and block sizes
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((Q + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   (D + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Compute L1 distances
    computeL1Distance<<<numBlocks, threadsPerBlock>>>(d_documents, d_queries, d_distances, D, N, Q);
    hipError_t err_dist = hipGetLastError();
    if (err_dist != hipSuccess) {
        std::cerr << "Failed to launch computeL1Distance kernel: " << hipGetErrorString(err_dist) << std::endl;
        return -1;
    }

    // Sum over the last dim
    dim3 blockDim(D);  // Ensure blockDim does not exceed 512
    dim3 gridDim(1, N, Q);
    size_t sharedMemSize = D * sizeof(float);
    sumOverLastDimKernel<<<gridDim, blockDim, sharedMemSize>>>(d_distances, d_results, D, N, Q);
    hipError_t err_sum = hipGetLastError();
    if (err_sum != hipSuccess) {
        std::cerr << "Failed to launch sumOverLastDimKernel kernel: " << hipGetErrorString(err_sum) << std::endl;
        return -1;
    }

    // Copy the result back to host
    hipMemcpy(h_results, d_results, Q * N * sizeof(float), hipMemcpyDeviceToHost);
    int* h_sorted_indices = argsort(h_results, Q, N);
    
    // Verification
#if DEBUG
    // Allocate memory
    float *h_distances_cpu = (float *)malloc(Q * N * D * sizeof(float));
    float *h_results_cpu = (float *)malloc(Q * N * sizeof(float));

    // Perform the same operations on the CPU
    computeL1DistanceCPU(h_documents, h_queries, h_distances_cpu, D, N, Q);
    sumOverLastDim(h_distances_cpu, h_results_cpu, D, N, Q);
    int* h_sorted_indices_cpu = argsort(h_results_cpu, Q, N);

    // Verify the distances by comparing the GPU and CPU results
    printf("\nVerifying distance computation...\n");
    for (int q = 0; q < Q; ++q) {
        float totalError = 0.0;
        for (int i = 0; i < N; ++i) {
            int index = q * N + i;
            totalError += h_results[index] - h_results_cpu[index];
        }
        float avgError = totalError / N;
        if (avgError > 1e-3)
            printf("Avg error for query %d: %f\n", q, avgError);
    }
    
    // Verify the sorting by comparing the GPU and CPU results
    printf("\nVerifying sorting...\n");
    for (int q = 0; q < Q; ++q) {
        float totalError = 0.0;
        for (int i = 0; i < N; ++i) {
            int index = q * N + i;
            totalError += h_sorted_indices[index] - h_sorted_indices_cpu[index];
        }
        float avgError = totalError / N;
        if (avgError > 1e-3)
            printf("Avg error for query %d: %f\n", q, avgError);
    }

    // Deallocate memory
    free(h_distances_cpu);
    free(h_results_cpu);
    free(h_sorted_indices_cpu);
    
#endif

    // Print a few results
    printf("\nTop documents for queries\n");
    printMatrix(h_sorted_indices, Q, N, Q, K);

    // Clean up memory
    hipFree(d_documents);
    hipFree(d_queries);
    hipFree(d_distances);
    hipFree(d_results);

    free(h_documents);
    free(h_queries);
    free(h_results);
    free(h_sorted_indices);

    return 0;
}
