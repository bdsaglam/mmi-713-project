#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath> // For fabsf
#include <iostream>
#include <stdlib.h>
#include "cli.h"
#include "common.h"
#include "constants.h"
#include "sorting.h"
#include "knn.h"
#include "knn.cu"


int main(int argc, char *argv[]) {
    Params params;
    parseCommandLine(argc, argv, params);

    int N = params.N;
    int Q = params.Q;

    // Allocate host memory
    float *h_documents = (float *)malloc(N * D * sizeof(float));
    float *h_queries = (float *)malloc(Q * D * sizeof(float));
    int *h_indices = (int *)malloc(Q * N * sizeof(int )); // Indices array to store the output of kSelectKernel

    // Initialize data with random values
    // srand(time(NULL));
    randomInit(h_documents, N, D);
    randomInit(h_queries, Q, D);

    // Mark start time
    clock_t start = clock();

    // Allocate device memory
    float *d_documents, *d_queries, *d_distances, *d_agg_distances;
    int *d_indices;
    hipMalloc(&d_documents, N * D * sizeof(float));
    hipMalloc(&d_queries, Q * D * sizeof(float));
    hipMalloc(&d_distances, Q * N * D * sizeof(float));
    hipMalloc(&d_agg_distances, Q * N * sizeof(float));
    hipMalloc(&d_indices, Q * N * sizeof(int )); // Device memory for indices

    // Copy data from host to device
    hipMemcpy(d_documents, h_documents, N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_queries, h_queries, Q * D * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid and block sizes
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((Q + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   (D + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Compute L1 distances
    computeL1DistanceKernel<<<numBlocks, threadsPerBlock>>>(d_documents, d_queries, d_distances, D, N, Q);
    hipError_t err_dist = hipGetLastError();
    if (err_dist != hipSuccess) {
        std::cerr << "Failed to launch computeL1DistanceKernel: " << hipGetErrorString(err_dist) << std::endl;
        return -1;
    }

    // Sum over the last dim
    dim3 blockDim(D);
    dim3 gridDim(1, N, Q);
    size_t sharedMemSize = D * sizeof(float);
    sumOverLastDimKernel<<<gridDim, blockDim, sharedMemSize>>>(d_distances, d_agg_distances, D, N, Q);
    hipError_t err_sum = hipGetLastError();
    if (err_sum != hipSuccess) {
        std::cerr << "Failed to launch sumOverLastDimKernel: " << hipGetErrorString(err_sum) << std::endl;
        return -1;
    }
      
    // Select k smallest elements
    int kSelectThreadsPerBlock = 1024;
    int kSelectBlocksPerGrid = (Q + kSelectThreadsPerBlock - 1) / kSelectThreadsPerBlock;
    kSelectKernel<<<kSelectBlocksPerGrid, kSelectThreadsPerBlock>>>(d_agg_distances, d_indices, Q, N, K);
    
    // Copy the sorted indices back to the host
    hipMemcpy(h_indices, d_indices, Q * N * sizeof(int ), hipMemcpyDeviceToHost);

    // Measure elapsed time
    clock_t end = clock();
    double elapsed_time_ms = 1000 * (double)(end - start) / CLOCKS_PER_SEC;
    printf("Elapsed time: %f ms\n", elapsed_time_ms);
    
    // Print results
    printResults(h_indices, Q, N, K);

    // Clean up memory
    hipFree(d_documents);
    hipFree(d_queries);
    hipFree(d_distances);
    hipFree(d_agg_distances);
    hipFree(d_indices);

    free(h_documents);
    free(h_queries);
    free(h_indices);

    return 0;
}
